
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void whoami(void) {
    int block_id =
        blockIdx.x +    // apartment number
        blockIdx.y * gridDim.x +    // floor number in this building
        blockIdx.z * gridDim.x * gridDim.y;   // building number in this city

    int block_offset =
        block_id * // times our apartment number
        blockDim.x * blockDim.y * blockDim.z; // total threads per block (people per apartment)

    int thread_offset = 
        threadIdx.x +  
        threadIdx.y * blockDim.x +
        threadIdx.z * blockDim.x * blockDim.y;

    int id = block_offset + thread_offset; // global person id in the entire apartment complex

    printf("%04d | Block(%d %d %d) = %3d | Thread(%d %d %d) = %3d\n",
        id,
        blockIdx.x, blockIdx.y, blockIdx.z, block_id,
        threadIdx.x, threadIdx.y, threadIdx.z, thread_offset);
}

int main(int argc, char **argv) {

    //dimensions of the grid
    const int b_x = 2, b_y = 3, b_z = 4;
    //dimensions of each block
    const int t_x = 4, t_y = 4, t_z = 4; // the max warp size is 32
    // so we will get 2 warps of 32 threads per block

    int blocks_per_grid = b_x * b_y * b_z;
    int threads_per_block = t_x * t_y * t_z;

    printf("%d blocks/grid\n", blocks_per_grid);
    printf("%d threads/block\n", threads_per_block);
    printf("%d total threads\n", blocks_per_grid * threads_per_block);

    dim3 blocksPerGrid(b_x, b_y, b_z); // 3d cube of shape 2*3*4 = 24
    dim3 threadsPerBlock(t_x, t_y, t_z); // 3d cube of shape 4*4*4 = 64

    whoami<<<blocksPerGrid, threadsPerBlock>>>();
    hipDeviceSynchronize();
}